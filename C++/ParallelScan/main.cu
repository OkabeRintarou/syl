#include "hip/hip_runtime.h"
#include "gpu_scan.cuh"
#include "scan.h"
#include "timer.h"
#include "util.h"
#include <iostream>
#include <memory>
#include <algorithm>
#include <omp.h>

static constexpr int ITER_COUNT = 5;

template<typename T>
static void verify(const T *expect, const T *actual, const int n) {
    for (int i = 0; i < n; i++) {
        if (actual[i] != expect[i]) {
            fprintf(stderr, "incorrect verify: [%d] expect: %d, actual: %d\n",
                    i, expect[i], actual[i]);
            break;
        }
    }
}

template<typename T>
static void prepare_cuda_scan(T *cpu_in, int n, T *&gpu_in, T *&gpu_out) {
    const int bytes = sizeof(T) * n;

    hipMalloc(&gpu_in, bytes);
    hipMalloc(&gpu_out, bytes);
    hipMemcpy(gpu_in, cpu_in, bytes, hipMemcpyHostToDevice);
}

template<typename T>
static T *test_cuda_scan_version_1(T *cpu_in, int n) {
    T *gpu_in, *gpu_out;
    prepare_cuda_scan(cpu_in, n, gpu_in, gpu_out);
    cuda_scan_version_1<<<1, 512, sizeof(int) * n>>>(gpu_out, gpu_in, n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "cuda scan version 1, error %d:%s\n",
                static_cast<int>(err),
                hipGetErrorString(err));
        hipFree(gpu_in);
        hipFree(gpu_out);
        return nullptr;
    }

    T *cpu_out = new T[n];
    hipMemcpy(cpu_out, gpu_out, sizeof(T) * n, hipMemcpyDeviceToHost);

    hipFree(gpu_in);
    hipFree(gpu_out);
    return cpu_out;
}

template<typename T>
static T *test_cuda_scan_avoid_bank_conflict(T *cpu_in, int n) {
    T *gpu_in, *gpu_out;
    prepare_cuda_scan(cpu_in, n, gpu_in, gpu_out);
    cuda_scan_avoid_bank_conflict<<<1, 512, sizeof(T) * (n + 32)>>>(gpu_out, gpu_in, n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "cuda scan version 1, error %d:%s\n",
                static_cast<int>(err),
                hipGetErrorString(err));
        hipFree(gpu_in);
        hipFree(gpu_out);
        return nullptr;
    }

    T *cpu_out = new T[n];
    hipMemcpy(cpu_out, gpu_out, sizeof(T) * n, hipMemcpyDeviceToHost);

    hipFree(gpu_in);
    hipFree(gpu_out);
    return cpu_out;
}

template<typename T>
static T *test_cuda_scan(T *cpu_in, int n) {
    T *cpu_out = new T[n];
    cuda_scan(cpu_out, cpu_in, n);
    return cpu_out;
}

int main() {
    for (int i = 0; i < 100; i++) {
        int size = 10243212;
        int *input = gen_int(size);
        int *output1 = new int[size];
        serial_scan(output1, input, size);
        int *output2 = test_cuda_scan(input, size);
        verify(output1, output2, size);
        delete []output1;
        delete []output2;
    }

    return 0;
}
